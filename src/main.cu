#include "hip/hip_runtime.h"
#define BLOCK_SIZE 256
#define DOWNSCALE_MAX_FRAME_HIGHT 400

#include <librealsense2/rs.hpp> // Include RealSense Cross Platform API
#include <librealsense2/rs_advanced_mode.hpp>
#include <librealsense2/rsutil.h>
#include <opencv2/opencv.hpp> // Include OpenCV API
#include <chrono>
#include "util.hpp"
#include "salient.cuh"

__global__ void draw_foreground(int N, uint8_t *out_rgb, const float *probabilities, const uint8_t *in_color)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    if (probabilities[idx] > 0.5f)
    {
        float Y = ((float)in_color[idx * 2]) - 16.0f;
        float Cb = ((float)in_color[(idx - idx % 2) * 2 + 1]) - 128.0f;
        float Cr = ((float)in_color[(idx - idx % 2) * 2 + 3]) - 128.0f;

        out_rgb[idx * 3 + 0] = (uint8_t)__float2int_rd(max(0.0f, min(255.0f, 1.163999557f * Y + 2.017999649f * Cb)));
        out_rgb[idx * 3 + 1] = (uint8_t)__float2int_rd(max(0.0f, min(255.0f, 1.163999557f * Y - 0.812999725f * Cr - 0.390999794f * Cb)));
        out_rgb[idx * 3 + 2] = (uint8_t)__float2int_rd(max(0.0f, min(255.0f, 1.163999557f * Y + 1.595999718f * Cr)));
    }
    else
    {
        out_rgb[idx * 3 + 0] = 0;
        out_rgb[idx * 3 + 1] = 150;
        out_rgb[idx * 3 + 2] = 0;
    }
}

int main(int argc, char *argv[])
try
{
    using namespace cv;
    using namespace rs2;

    // Start the camera
    pipeline pipe;
    rs2::config config;
    config.enable_stream(RS2_STREAM_DEPTH, 1280, 720, RS2_FORMAT_Z16, 30);
    // config.enable_stream(RS2_STREAM_COLOR, 1280, 720, RS2_FORMAT_YUYV, 30);
    config.enable_stream(RS2_STREAM_COLOR, 1920, 1080, RS2_FORMAT_YUYV, 30);
    // config.enable_stream(RS2_STREAM_DEPTH, 848, 480, RS2_FORMAT_Z16, 90)
    // config.enable_stream(RS2_STREAM_COLOR, 848, 480, RS2_FORMAT_YUYV, 60)

    auto selection = pipe.start(config);
    auto dev = selection.get_device();
    auto sensor = dev.first<rs2::depth_sensor>();

    // load json preset for high accuracy mode
    auto dev_adv = rs400::advanced_mode::advanced_mode(dev);
    std::ifstream settingsJSONFile("preset.json");
    std::string settingsJSON((std::istreambuf_iterator<char>(settingsJSONFile)), std::istreambuf_iterator<char>());
    dev_adv.load_json(settingsJSON);

    // get dimension (to be sure)
    auto depth_stream = selection.get_stream(RS2_STREAM_DEPTH).as<rs2::video_stream_profile>();
    auto color_stream = selection.get_stream(RS2_STREAM_COLOR).as<rs2::video_stream_profile>();
    auto color_W = color_stream.width();
    auto color_H = color_stream.height();
    auto color_N(color_W * color_H);
    auto W = color_W;
    auto H = color_H;
    auto downsample_ratio = 1;
    while (H > DOWNSCALE_MAX_FRAME_HIGHT)
    {
        downsample_ratio <<= 1;
        H >>= 1;
        W >>= 1;
    }

    auto rs2_depth_intr = depth_stream.get_intrinsics();
    auto rs2_color_intr = color_stream.get_intrinsics();
    auto rs2_color_to_depth = color_stream.get_extrinsics_to(depth_stream);

    const salient::CameraIntrinsics depthIntr = {
        rs2_depth_intr.width,
        rs2_depth_intr.height,
        rs2_depth_intr.ppx,
        rs2_depth_intr.ppy,
        rs2_depth_intr.fx,
        rs2_depth_intr.fy};

    const salient::CameraIntrinsics colorIntr = {
        rs2_color_intr.width,
        rs2_color_intr.height,
        rs2_color_intr.ppx,
        rs2_color_intr.ppy,
        rs2_color_intr.fx,
        rs2_color_intr.fy};

    // rely on the fact that we have the same representation as librealsense
    const salient::CameraExtrinsics color2depth(*reinterpret_cast<salient::CameraExtrinsics *>(&rs2_color_to_depth));

    // Select the GPU.
    // The idea is to select a secondary, less powerfull GPU for this, so that it does not interfere with
    // the main user activity (such as playing VR).
    hipSetDevice(0);

    hipStream_t mainStream;
    hipStreamCreate(&mainStream);

    // original image from the color camera
    uint8_t *yuyvGPU = nullptr;
    hipMalloc((void **)&yuyvGPU, sizeof(uint8_t) * color_W * color_H * 2);
    cudaErrorCheck(nullptr);

    // transformed RGB image with foreground mask applied
    uint8_t *rgbGPU = nullptr;
    hipMalloc((void **)&rgbGPU, sizeof(uint8_t) * color_W * color_H * 3);
    cudaErrorCheck(nullptr);

    // how to access color data at every pixel position.
    auto getFeature = [yuyvGPU, color_W] __device__(const int i, const int j, float *out_feature) {
        const int base_off = i + j * color_W;
        out_feature[0] = (float)yuyvGPU[base_off * 2];
        out_feature[1] = 0.5f * (float)yuyvGPU[(base_off - i % 2) * 2 + 1];
        out_feature[2] = 0.5f * (float)yuyvGPU[(base_off - i % 2) * 2 + 3];
    };
    salient::RealSalient<3, 7, decltype(getFeature)> realSalient(
        mainStream, depthIntr, colorIntr, color2depth, downsample_ratio, sensor.get_depth_scale(), getFeature);

    const auto window_name = "real-salient";
    namedWindow(window_name, WINDOW_AUTOSIZE);

    double frame_cap = 90, fps = frame_cap;
    auto frame_avg_time = 1.0 / fps;
    auto frame_start_time = std::chrono::high_resolution_clock::now();
    auto frame_stop_time = frame_start_time;
    // auto frame_cap_ms = 1000 / frame_cap;
    auto ema_alpha = 0.1;

    // Skips some frames to allow for auto-exposure stabilization
    for (int i = 0; i < 10; i++)
        pipe.wait_for_frames();

    Mat3b foreground;
    foreground.create(Size(color_W, color_H));

    for (int frame_number = 0; waitKey(1) < 0 && getWindowProperty(window_name, WND_PROP_AUTOSIZE) >= 0; frame_number++)
    {
        frame_stop_time = std::chrono::high_resolution_clock::now();
        auto frame_time = std::chrono::duration_cast<std::chrono::microseconds>(frame_stop_time - frame_start_time);
        frame_avg_time = frame_avg_time * (1 - ema_alpha) + frame_time.count() * ema_alpha / 1000000;
        if (frame_number % std::max(1, (int)std::round(fps / 4.0)) == 0)
            fps = 1 / frame_avg_time;
        frame_start_time = frame_stop_time;

        frameset data = pipe.wait_for_frames();

        // copy the color frame, so that getFeature gets the actual color data.
        hipMemcpyAsync(yuyvGPU, data.get_color_frame().get_data(), sizeof(uint8_t) * color_N * 2, hipMemcpyHostToDevice, mainStream);
        cudaErrorCheck(mainStream);

        // load frames to gpu and preprocess
        realSalient.processFrames(
            (const uint16_t *)data.get_depth_frame().get_data(),
            0.1f /* cutoff near distance in meters */,
            1.5f /* cutoff far distance in meters */,
            10 /* number of iterations in EM estimation algorithm for GMMs*/,
            5 /* number of passes in CRF inference */);

        draw_foreground<<<((color_N - 1) / BLOCK_SIZE + 1), BLOCK_SIZE, 0, mainStream>>>(color_N, rgbGPU, realSalient.probabilities, yuyvGPU);
        cudaErrorCheck(mainStream);

        hipMemcpyAsync(foreground.data, rgbGPU, sizeof(uint8_t) * color_W * color_H * 3, hipMemcpyDeviceToHost, mainStream);
        cudaErrorCheck(mainStream);

        // Show FPS
        cv::putText(foreground, string_format("FPS: %.1f", fps),
                    cv::Point(20, 50), cv::FONT_HERSHEY_SIMPLEX, 0.8, cv::Scalar(255, 255, 255), 2, cv::LINE_AA);

        imshow(window_name, foreground);
    }

    hipFree(rgbGPU);
    cudaErrorCheck(nullptr);
    hipFree(yuyvGPU);
    cudaErrorCheck(nullptr);

    return EXIT_SUCCESS;
}
catch (const rs2::error &e)
{
    std::cerr << "RealSense error calling " << e.get_failed_function() << "(" << e.get_failed_args() << "):\n    " << e.what() << std::endl;
    return EXIT_FAILURE;
}
catch (const std::exception &e)
{
    std::cerr << e.what() << std::endl;
    return EXIT_FAILURE;
}
